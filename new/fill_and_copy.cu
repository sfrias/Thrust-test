#include <iostream>
#include <cmath>

#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>

using std::cout;
using std::cerr;
using std::endl;
using std::flush;

void fill_test() {
	cout << "Fill test ... " << flush;

	const int ARRAY_SIZE = 1000;

	double *mA;

	hipMallocManaged(&mA, sizeof(double) * ARRAY_SIZE);

	for (int i = 0; i < ARRAY_SIZE; i++)
		mA[i] = 0.0;

	thrust::fill(thrust::cuda::par, mA, mA + ARRAY_SIZE, 9.0);
	hipDeviceSynchronize();

	bool correct = true;
	for (int i = 0; i < ARRAY_SIZE; i++)
		if (mA[i] != 9.0) {
			correct = false;
			break;
		}

	hipFree(mA);

	if (correct)
		cout << "OK" << endl;
	else
		cout << "Failed" << endl;
}

void copy_test() {
	cout << "Copy test ... " << flush;

	const int ARRAY_SIZE = 1000;

	double *mA, *mB;

	hipMallocManaged(&mA, sizeof(double) * ARRAY_SIZE);
	hipMallocManaged(&mB, sizeof(double) * ARRAY_SIZE);

	for (int i = 0; i < ARRAY_SIZE; i++)
		mA[i] = 1.0 * (i+1);

	thrust::copy(thrust::cuda::par, mA, mA + ARRAY_SIZE, mB);
	hipDeviceSynchronize();

	bool correct = true;
	for (int i = 0; i < ARRAY_SIZE; i++)
		if (mB[i] != 1.0 * (i + 1)) {
			correct = false;
			break;
		}

	hipFree(mA);
	hipFree(mB);

	if (correct)
		cout << "OK" << endl;
	else
		cout << "Failed" << endl;

}

int main(int argc, char **argv) 
{
	fill_test();
	copy_test();
	return 0;
}
