#include <iostream>
#include <cmath>

#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/gather.h>
#include <thrust/logical.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>

const int ARRAY_SIZE = 1000;

enum Method {
	RAW,
	WRAPPED
};

// ------------------------------------------------------------------------------------

bool check_transform(double* hA)
{
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (hA[i] != - 1.0 * (i + 1))
			return false;
	}

	return true;
}

bool transform_test(Method method)
{
	double *hA;
	hA = (double *) malloc(ARRAY_SIZE * sizeof(double));
	for (int i = 0; i < ARRAY_SIZE; i++)
		hA[i] = 1.0 * (i + 1);

	double* dA;
	hipMalloc((void **) &dA, ARRAY_SIZE * sizeof(double));
	hipMemcpy(dA, hA, sizeof(double) * ARRAY_SIZE, hipMemcpyHostToDevice);

	switch (method) {
	case RAW:
		{
			thrust::transform(thrust::cuda::par, dA, dA + ARRAY_SIZE, dA, thrust::negate<double>());
			break;
		}
	case WRAPPED:
		{
			thrust::device_ptr<double> wdA = thrust::device_pointer_cast(dA);
			thrust::transform(wdA, wdA + ARRAY_SIZE, wdA, thrust::negate<double>());
			break;
		}
	}

	hipMemcpy(hA, dA, sizeof(double) * ARRAY_SIZE, hipMemcpyDeviceToHost);
	bool result = check_transform(hA);

	hipFree(dA);
	free(hA);

	return result;
}

// ------------------------------------------------------------------------------------

bool check_transform_if(double* hA)
{
	for (int i = 0; i < (ARRAY_SIZE >> 1); i++) {
		if (hA[i] != 2.0 * (i + 1))
			return false;
	}

	for (int i = (ARRAY_SIZE >> 1); i < ARRAY_SIZE; i++) {
		if (hA[i] != 1.0 * (i + 1))
			return false;
	}

	return true;
}

bool transform_if_test(Method method)
{
	double* hA;
	double* hB;
	int*    h_stencil;
	hA = (double *) malloc(ARRAY_SIZE * sizeof(double));
	hB = (double *) malloc(ARRAY_SIZE * sizeof(double));
	h_stencil = (int *) malloc(ARRAY_SIZE * sizeof(int));
	for (int i = 0; i < ARRAY_SIZE; i++)
		hB[i] = hA[i] = 1.0 * (i + 1);

	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (i < (ARRAY_SIZE >> 1))
			h_stencil[i] = 1;
		else
			h_stencil[i] = 0;
	}

	double* dA;
	double* dB;
	int*    d_stencil;
	hipMalloc((void **) &dA, ARRAY_SIZE * sizeof(double));
	hipMalloc((void **) &dB, ARRAY_SIZE * sizeof(double));
	hipMalloc((void **) &d_stencil, ARRAY_SIZE * sizeof(int));
	hipMemcpy(dA, hA, sizeof(double) * ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(double) * ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_stencil, h_stencil, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice);

	switch (method) {
	case RAW:
		{
			thrust::transform_if(thrust::cuda::par, dA, dA + ARRAY_SIZE, dB, d_stencil, dA, thrust::plus<double>(), thrust::identity<int>());
			break;
		}
	case WRAPPED:
		{
			thrust::device_ptr<double> wdA = thrust::device_pointer_cast(dA);
			thrust::device_ptr<double> wdB = thrust::device_pointer_cast(dB);
			thrust::device_ptr<int> wdS = thrust::device_pointer_cast(d_stencil);
			thrust::transform_if(wdA, wdA + ARRAY_SIZE, wdB, wdS, wdA, thrust::plus<double>(), thrust::identity<int>());
			break;
		}
	}

	hipMemcpy(hA, dA, sizeof(double) * ARRAY_SIZE, hipMemcpyDeviceToHost);
	bool result = check_transform_if(hA);

	hipFree(dA);
	hipFree(dB);
	hipFree(d_stencil);
	free(hA);
	free(hB);
	free(h_stencil);

	return result;
}

// ------------------------------------------------------------------------------------

bool check_sequence(double* hA)
{
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (hA[i] != 1.0 * i)
			return false;
	}

	return true;
}

bool sequence_test(Method method) {
	double* hA;
	hA = (double *) malloc(sizeof(double) * ARRAY_SIZE);
	for (int i = 0; i < ARRAY_SIZE; i++)
		hA[i] = 0.0;

	double* dA;
	hipMalloc((void **) &dA, sizeof(double) * ARRAY_SIZE);

	switch (method) {
	case RAW:
		{
			thrust::sequence(thrust::cuda::par, dA, dA + ARRAY_SIZE);
			break;
		}
	case WRAPPED:
		{
			thrust::device_ptr<double> wdA = thrust::device_pointer_cast(dA);
			thrust::sequence(wdA, wdA + ARRAY_SIZE);
			break;
		}
	}

	hipMemcpy(hA, dA, sizeof(double) * ARRAY_SIZE, hipMemcpyDeviceToHost);
	bool result = check_sequence(hA);

	free(hA);
	hipFree(dA);

	return result;
}


// ------------------------------------------------------------------------------------

bool check_tabulate(double* hA)
{
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (hA[i] != -1.0 * i)
			return false;
	}

	return true;
}

bool tabulate_test(Method method) {
	double* hA;
	hA = (double *) malloc(sizeof(double) * ARRAY_SIZE);

	double* dA;
	hipMalloc((void **) &dA, sizeof(double) * ARRAY_SIZE);

	switch (method) {
	case RAW:
		{
			thrust::tabulate(thrust::cuda::par, dA, dA + ARRAY_SIZE, thrust::negate<double>());
			break;
		}
	case WRAPPED:
		{
			thrust::device_ptr<double> wdA = thrust::device_pointer_cast(dA);
			thrust::tabulate(wdA, wdA + ARRAY_SIZE, thrust::negate<double>());
			break;
		}
	}

	hipMemcpy(hA, dA, sizeof(double) * ARRAY_SIZE, hipMemcpyDeviceToHost);
	bool result = check_tabulate(hA);

	free(hA);
	hipFree(dA);

	return result;
}

// ------------------------------------------------------------------------------------

int main(int argc, char **argv)
{
	std::cout << "Transform DR ... " << std::flush << transform_test(RAW) << std::endl;
	std::cout << "Transform DW ... " << std::flush << transform_test(WRAPPED) << std::endl;

	std::cout << "Transform_if DR ... " << std::flush << transform_if_test(RAW) << std::endl;
	std::cout << "Transform_if DW ... " << std::flush << transform_if_test(WRAPPED) << std::endl;

	std::cout << "Sequence DR ... " << std::flush << sequence_test(RAW) << std::endl;
	std::cout << "Sequence DW ... " << std::flush << sequence_test(WRAPPED) << std::endl;

	std::cout << "Tabulate DR ... " << std::flush << tabulate_test(RAW) << std::endl;
	std::cout << "Tabulate DW ... " << std::flush << tabulate_test(WRAPPED) << std::endl;

	return 0;
}
